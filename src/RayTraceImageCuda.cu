#include "hip/hip_runtime.h"
#include "RayTraceStructures.h"
#include "common/RayTraceImageHelper.h"
#include "utilities/RayUtilities.h"
#include <hip/hip_runtime.h>


// Check for errors
#define CUDA_CHECK()                                    \
    do {                                                \
        if ( hipPeekAtLastError() != hipSuccess ) {   \
            hipError_t error = hipGetLastError();     \
            printf("cuda error: %i\n",error);           \
            printf("   %s\n",hipGetErrorString(error)); \
            printf("   line: %i\n",(int)__LINE__);      \
            printf("   file: %s\n",__FILE__);           \
            exit(-1);                                   \
        }                                               \
    } while (0)


#define CUDA_PRINT_FUNCTION( fun )                      \
    do {                                                \
        hipFuncAttributes attr;                        \
        hipFuncGetAttributes(&attr,reinterpret_cast<const void*>(fun));               \
        printf("%s:\n",#fun);                           \
        printf("  version = %i\n",attr.binaryVersion);  \
        printf("  ptx = %i\n",attr.ptxVersion);         \
        printf("  constSize = %i\n",attr.constSizeBytes); \
        printf("  localSize = %i\n",attr.localSizeBytes); \
        printf("  sharedSize = %i\n",attr.sharedSizeBytes); \
        printf("  maxThreads = %i\n",attr.maxThreadsPerBlock); \
        printf("  numRegs = %i\n",attr.numRegs);        \
    } while (0)


// Atomic add operation for double
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, 
                        __double_as_longlong(val + 
                        __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}


// Get the globally unique thread id
__device__ int getGlobalIdx3D()
{
	int blockId = blockIdx.x 
			 + blockIdx.y * gridDim.x 
			 + gridDim.x * gridDim.y * blockIdx.z; 
	int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
			  + (threadIdx.z * (blockDim.x * blockDim.y))
			  + (threadIdx.y * blockDim.x)
			  + threadIdx.x;
	return threadId;
}


// Kernel that executes on the CUDA device
__global__
__launch_bounds__(128,8)    // Set bounds to limit the number of registers
void RayTraceImageCudaKernel( int N, int nx, int ny, int na, int nb, int nv, 
    const double *x, const double *y, const double *a, const double *b,
    double dx, double dy, double dz, double da, double db, const double *dv,
    const RayTrace::ray_gain_struct* gain, const RayTrace::ray_seed_struct* seed,
    int method, int N_rays, const ray_struct *rays, double scale,
    double *image, double *I_ang ) 
{
    int idx = getGlobalIdx3D();
    if ( idx<N_rays ) {
        const ray_struct ray = rays[idx];
        double Iv[K_MAX];
        ray_struct ray2;
        int error = RayTrace_calc_ray( ray, N, dz, gain, seed, nv, method, Iv, ray2 );
        if ( error!=0 ) {
            //failed_rays.push_back(ray);
            //set_bit(-error,failure_code);
        } else {
            if ( method == 1 ) {
                // We are propagating backward, use ray for the cell updates
                ray2 = ray;
            } else {
                // We are propagating forward, use ray2 for the cell updates
                // Note: The sign of the angle is reversed with respect to the euv_beam
                ray2.a = -ray2.a;
                ray2.b = -ray2.b;
                if ( ray2.y<0.0 && y[0]>=0.0 ) {
                    // We need to change the sign of y
                    ray2.y = -ray2.y;
                }
            }
            // Get the indicies to the cells in image and I_ang
            // Note: do not replace these lines with findindex, we need to be able to return 0 for the index
            int i1 = static_cast<int>( findfirstsingle( x, nx, ray2.x-0.5*dx ) );
            int i2 = static_cast<int>( findfirstsingle( y, ny, ray2.y-0.5*dy ) );
            int i3 = static_cast<int>( findfirstsingle( a, na, ray2.a-0.5*da ) );
            int i4 = static_cast<int>( findfirstsingle( b, nb, ray2.b-0.5*db ) );
            if ( ray2.x<x[0]-0.5*dx || ray2.x>x[nx-1]+0.5*dx )
                i1 = -1;        // The ray's z position is out of the range of image 
            if ( ray2.y<y[0]-0.5*dy || ray2.y>y[ny-1]+0.5*dy )
                i2 = -1;        // The ray's y position is out of the range of image 
            if ( -ray2.a<a[0]-0.5*da || -ray2.a>a[na-1]+0.5*da )
                i3 = -1;        // The ray's z angle is out of the range of I_ang 
            if ( -ray2.b<b[0]-0.5*db || -ray2.b>b[nb-1]+0.5*db )
                i4 = -1;        // The ray's y angle is out of the range of I_ang
            // Copy I_out into image 
            if (i1>=0 && i2>=0){ 
                double *Iv2 = &image[nv*(i1+i2*nx)];
                for (int iv=0; iv<nv; iv++)
                    atomicAdd(&Iv2[iv],Iv[iv]*scale);
            }
            // Copy I_out into I_ang 
            if (i3>=0 && i4>=0) {    
                double tmp = 0.0;
                for (int iv=0; iv<nv; iv++)
                    tmp += 2.0*dv[iv]*Iv[iv];
                atomicAdd(&I_ang[i3+i4*na],tmp);
            }
        }
    }
}


// Compute the block size to use
inline dim3 calcBlockSize( size_t N_blocks )
{
    dim3 block_size;
    if ( N_blocks < 65535 ) {
        block_size.x = N_blocks;
    } else {
        block_size.y = N_blocks/32768;
        block_size.x = N_blocks/block_size.y + (N_blocks%block_size.y == 0 ? 0:1);
    }
    return block_size;
}


// Create the image and call the cuda kernel
void RayTraceImageCudaLoop( int N, int nx, int ny, int na, int nb, int nv, 
    const double *x, const double *y, const double *a, const double *b,
    double dx, double dy, double dz, double da, double db, const double *dv,
    const RayTrace::ray_gain_struct* gain_in, const RayTrace::ray_seed_struct* seed_in,
    int method, const std::vector<ray_struct>& rays, double scale,
    double *image, double *I_ang, 
    unsigned int& failure_code, std::vector<ray_struct>& failed_rays ) 
{
    failure_code = 0;   // Need to track failures on GPU
    // Get device properties
    static int maxThreadsPerBlock = 0;
    if ( maxThreadsPerBlock == 0 ) {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr,reinterpret_cast<const void*>(RayTraceImageCudaKernel));
        maxThreadsPerBlock = attr.maxThreadsPerBlock;
        RAY_ASSERT(maxThreadsPerBlock>0);
        //CUDA_PRINT_FUNCTION(RayTraceImageCudaKernel);
    }    
    // place the ray gain and seed structures on the device
    const RayTrace::ray_gain_struct* gain = RayTrace::ray_gain_struct::copy_device( N, gain_in );
    const RayTrace::ray_seed_struct* seed = NULL;
    if ( seed_in!=NULL )
        seed = seed_in->copy_device();
    // Allocate device memory
    size_t N_rays = rays.size();
    double *x2, *y2, *a2, *b2, *dv2, *image2, *I_ang2;
    ray_struct *rays2;
    hipMalloc(&x2,nx*sizeof(double));
    hipMalloc(&y2,ny*sizeof(double));
    hipMalloc(&a2,na*sizeof(double));
    hipMalloc(&b2,nb*sizeof(double));
    hipMalloc(&dv2,nv*sizeof(double));
    hipMalloc(&image2,nx*ny*nv*sizeof(double));
    hipMalloc(&I_ang2,na*nb*sizeof(double));
    hipMalloc(&rays2,N_rays*sizeof(ray_struct));
    hipMemcpy(x2,x,nx*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(y2,y,ny*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(a2,a,na*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(b2,b,nb*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(dv2,dv,nv*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(rays2,&rays[0],N_rays*sizeof(ray_struct),hipMemcpyHostToDevice);
    hipMemset(image2,0,nx*ny*nv*sizeof(double));
    hipMemset(I_ang2,0,na*nb*sizeof(double));
    CUDA_CHECK();
    // Do calculation on device:
    size_t threads = maxThreadsPerBlock;
    size_t N_blocks = N_rays/threads + (N_rays%threads == 0 ? 0:1);
    dim3 block_size = calcBlockSize(N_blocks);
    block_size.x = N_rays/threads + (N_rays%threads == 0 ? 0:1);
    RayTraceImageCudaKernel <<< block_size,threads >>> (N,nx,ny,na,nb,nv,x2,y2,a2,b2,
        dx,dy,dz,da,db,dv2,gain,seed,method,N_rays,rays2,scale,image2,I_ang2);
    CUDA_CHECK();
    // Retrieve result from device and store it in host array
    hipMemcpy(image,image2,nx*ny*nv*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(I_ang,I_ang2,na*nb*sizeof(double),hipMemcpyDeviceToHost);
    CUDA_CHECK();
    // Cleanup
    hipFree(x2);
    hipFree(y2);
    hipFree(a2);
    hipFree(b2);
    hipFree(dv2);
    hipFree(rays2);
    hipFree(image2);
    hipFree(I_ang2);
    CUDA_CHECK();
    RayTrace::ray_gain_struct::free_device( N, gain_in, gain );
    RayTrace::ray_seed_struct::free_device( seed_in, seed );
}


// Copy ray_gain_struct to GPU
const RayTrace::ray_gain_struct* ray_gain_struct_copy_device_cuda( size_t N, const RayTrace::ray_gain_struct* arr )
{
    RayTrace::ray_gain_struct* host_ptr = new RayTrace::ray_gain_struct[N];
    for (size_t i=0; i<N; i++) {
        host_ptr[i].Nx = arr[i].Nx;
        host_ptr[i].Ny = arr[i].Ny;
        host_ptr[i].Nv = arr[i].Nv;
        hipMalloc(&host_ptr[i].x,arr[i].Nx*sizeof(double));
        hipMalloc(&host_ptr[i].y,arr[i].Ny*sizeof(double));
        hipMalloc(&host_ptr[i].n,arr[i].Nx*arr[i].Ny*sizeof(double));
        hipMalloc(&host_ptr[i].g0,arr[i].Nx*arr[i].Ny*sizeof(float));
        hipMalloc(&host_ptr[i].E0,arr[i].Nx*arr[i].Ny*sizeof(float));
        hipMalloc(&host_ptr[i].gv,arr[i].Nx*arr[i].Ny*arr[i].Nv*sizeof(float));
        hipMalloc(&host_ptr[i].gv0,arr[i].Nx*arr[i].Ny*sizeof(float));
        hipMemcpy(host_ptr[i].x,arr[i].x,arr[i].Nx*sizeof(double),hipMemcpyHostToDevice);
        hipMemcpy(host_ptr[i].y,arr[i].y,arr[i].Ny*sizeof(double),hipMemcpyHostToDevice);
        hipMemcpy(host_ptr[i].n,arr[i].n,arr[i].Nx*arr[i].Ny*sizeof(double),hipMemcpyHostToDevice);
        hipMemcpy(host_ptr[i].g0,arr[i].g0,arr[i].Nx*arr[i].Ny*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(host_ptr[i].E0,arr[i].E0,arr[i].Nx*arr[i].Ny*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(host_ptr[i].gv,arr[i].gv,arr[i].Nx*arr[i].Ny*arr[i].Nv*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(host_ptr[i].gv0,arr[i].gv0,arr[i].Nx*arr[i].Ny*sizeof(float),hipMemcpyHostToDevice);
    }
    RayTrace::ray_gain_struct* dev_ptr=NULL;
    hipMalloc(&dev_ptr,N*sizeof(RayTrace::ray_gain_struct));
    hipMemcpy(dev_ptr,host_ptr,N*sizeof(RayTrace::ray_gain_struct),hipMemcpyHostToDevice);
    for (size_t i=0; i<N; i++) {
        host_ptr[i].x = NULL;
        host_ptr[i].y = NULL;
        host_ptr[i].n = NULL;
        host_ptr[i].g0 = NULL;
        host_ptr[i].E0 = NULL;
        host_ptr[i].gv = NULL;
        host_ptr[i].gv0 = NULL;
    }
    delete [] host_ptr;
    CUDA_CHECK();
    return dev_ptr;
}
// Free ray_gain_struct from GPU
void ray_gain_struct_free_device_cuda( size_t N, const RayTrace::ray_gain_struct* dev_ptr )
{
    RayTrace::ray_gain_struct* host_ptr = new RayTrace::ray_gain_struct[N];
    hipMemcpy(host_ptr,dev_ptr,N*sizeof(RayTrace::ray_gain_struct),hipMemcpyDeviceToHost);
    for (size_t i=0; i<N; i++) {
        hipFree(host_ptr[i].x);
        hipFree(host_ptr[i].y);
        hipFree(host_ptr[i].n);
        hipFree(host_ptr[i].g0);
        hipFree(host_ptr[i].E0);
        hipFree(host_ptr[i].gv);
        hipFree(host_ptr[i].gv0);
        host_ptr[i].x = NULL;
        host_ptr[i].y = NULL;
        host_ptr[i].n = NULL;
        host_ptr[i].g0 = NULL;
        host_ptr[i].E0 = NULL;
        host_ptr[i].gv = NULL;
        host_ptr[i].gv0 = NULL;
    }
    hipFree((void*)dev_ptr);
    delete [] host_ptr;
    CUDA_CHECK();
}


// Copy ray_seed_struct to GPU
const RayTrace::ray_seed_struct* ray_seed_struct_copy_device_cuda( const RayTrace::ray_seed_struct& seed )
{
    RayTrace::ray_seed_struct* host_ptr = new RayTrace::ray_seed_struct();
    host_ptr->f0 = seed.f0;
    for (size_t i=0; i<5; i++) {
        host_ptr->dim[i] = seed.dim[i];
        hipMalloc(&host_ptr->x[i],seed.dim[i]*sizeof(double));
        hipMalloc(&host_ptr->f[i],seed.dim[i]*sizeof(double));
        hipMemcpy(host_ptr->x[i],seed.x[i],seed.dim[i]*sizeof(double),hipMemcpyHostToDevice);
        hipMemcpy(host_ptr->f[i],seed.f[i],seed.dim[i]*sizeof(double),hipMemcpyHostToDevice);
    }
    RayTrace::ray_seed_struct* dev_ptr=NULL;
    hipMalloc(&dev_ptr,sizeof(RayTrace::ray_seed_struct));
    hipMemcpy(dev_ptr,host_ptr,sizeof(RayTrace::ray_seed_struct),hipMemcpyHostToDevice);
    for (size_t i=0; i<5; i++) {
        host_ptr->x[i] = NULL;
        host_ptr->f[i] = NULL;
    }
    delete host_ptr;
    CUDA_CHECK();
    return dev_ptr;
}
// Free ray_seed_struct from GPU
void ray_seed_struct_free_device_cuda( const RayTrace::ray_seed_struct* dev_ptr )
{
    RayTrace::ray_seed_struct* host_ptr = new RayTrace::ray_seed_struct;
    hipMemcpy(host_ptr,dev_ptr,sizeof(RayTrace::ray_seed_struct),hipMemcpyDeviceToHost);
    for (size_t i=0; i<5; i++) {
        hipFree(host_ptr->x[i]);
        hipFree(host_ptr->f[i]);
        host_ptr->x[i] = NULL;
        host_ptr->f[i] = NULL;
    }
    hipFree((void*)dev_ptr);
    delete host_ptr;
    CUDA_CHECK();
}


